#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>
#include <stdlib.h>

#define MPICHECK(cmd) do {                          \
  int e = cmd;                                      \
  if( e != MPI_SUCCESS ) {                          \
    printf("Failed: MPI error %s:%d '%d'\n",        \
        __FILE__,__LINE__, e);   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t re = cmd;                             \
  if (re!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(re));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


static uint64_t getHostHash(const char* string) {
  // Based on DJB2a, result = result * 33 ^ char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++){
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}


static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i=0; i< maxlen; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        return;
    }
  }
}


int main(int argc, char* argv[])
{
  int size = 2*1024*1024;
  // int size = 32*1024*1024;


  int myRank, nRanks, localRank = 0;


  //initializing MPI
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));


  //calculating localRank based on hostname which is used in selecting a GPU
  uint64_t hostHashs[nRanks];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);
  MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p=0; p<nRanks; p++) {
     if (p == myRank) break;
     if (hostHashs[p] == hostHashs[myRank]) localRank++;
  }

  printf("The local rank is: %d\n", localRank);


  ncclUniqueId id;
  ncclComm_t comm;
  float **sendbuff, **recvbuff;
  hipStream_t s;


  //get NCCL unique ID at rank 0 and broadcast it to all others
  if (myRank == 0) ncclGetUniqueId(&id);
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));

  sendbuff = (float**)malloc(nRanks * sizeof(float*));
  recvbuff = (float**)malloc(nRanks * sizeof(float*));

  //picking a GPU based on localRank, allocate device buffers
  CUDACHECK(hipSetDevice(localRank));

  for (int r = 0; r < nRanks; r++) {
    CUDACHECK(hipMalloc(&sendbuff[r], size * sizeof(float)));
    CUDACHECK(hipMalloc(&recvbuff[r], size * sizeof(float)));
  }
  
  for (int r = 0; r < nRanks; r++) {
    CUDACHECK(hipMemset(sendbuff[r], 0, size * sizeof(float)));
    CUDACHECK(hipMemset(recvbuff[r], 0, size * sizeof(float)));
  }

  CUDACHECK(hipStreamCreate(&s));


  //initializing NCCL
  NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));


  //communicating using NCCL
  ncclGroupStart();
  for (int r = 0; r < nRanks; r++) {
      NCCLCHECK(ncclSend((const void*)sendbuff[r], size, ncclFloat, r, comm, s));
      NCCLCHECK(ncclRecv((void*)recvbuff[r], size, ncclFloat, r, comm, s));
  }
  ncclGroupEnd();

  //completing NCCL operation by synchronizing on the CUDA stream
  CUDACHECK(hipStreamSynchronize(s));


  //free device buffers
  for (int r = 0; r < nRanks; r++) {
    CUDACHECK(hipFree(sendbuff[r]));
    CUDACHECK(hipFree(recvbuff[r]));
  }
  free(sendbuff);
  free(recvbuff);


  //finalizing NCCL
  ncclCommDestroy(comm);


  //finalizing MPI
  MPICHECK(MPI_Finalize());


  printf("[MPI Rank %d] Success \n", myRank);
  
  return 0;
}
