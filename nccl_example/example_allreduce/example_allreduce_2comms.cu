#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>
#include <stdlib.h>

#define MPICHECK(cmd) do {                          \
  int e = cmd;                                      \
  if( e != MPI_SUCCESS ) {                          \
    printf("Failed: MPI error %s:%d '%d'\n",        \
        __FILE__,__LINE__, e);   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


static uint64_t getHostHash(const char* string) {
  // Based on DJB2a, result = result * 33 ^ char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++){
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}


static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i=0; i< maxlen; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        return;
    }
  }
}


int main(int argc, char* argv[])
{
  int size = 2*1024*1024;
  // int size = 32*1024*1024;


  int myRank, nRanks, localRank = 0;


  //initializing MPI
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));


  //calculating localRank based on hostname which is used in selecting a GPU
  uint64_t hostHashs[nRanks];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);
  MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p=0; p<nRanks; p++) {
     if (p == myRank) break;
     if (hostHashs[p] == hostHashs[myRank]) localRank++;
  }

  printf("The local rank is: %d\n", localRank);


  ncclUniqueId id_0;
  ncclUniqueId id_1;
  ncclComm_t comm_0;
  ncclComm_t comm_1;
  float *sendbuff_0, *recvbuff_0;
  float *sendbuff_1, *recvbuff_1;
  hipStream_t s_0;
  hipStream_t s_1;


  //get NCCL unique ID at rank 0 and broadcast it to all others
  if (myRank == 0) ncclGetUniqueId(&id_0);
  if (myRank == 0) ncclGetUniqueId(&id_1);
  MPICHECK(MPI_Bcast((void *)&id_0, sizeof(id_0), MPI_BYTE, 0, MPI_COMM_WORLD));
  MPICHECK(MPI_Bcast((void *)&id_1, sizeof(id_1), MPI_BYTE, 0, MPI_COMM_WORLD));


  //picking a GPU based on localRank, allocate device buffers
  CUDACHECK(hipSetDevice(localRank));
  CUDACHECK(hipMalloc(&sendbuff_0, size * sizeof(float)));
  CUDACHECK(hipMalloc(&recvbuff_0, size * sizeof(float)));
  CUDACHECK(hipMalloc(&sendbuff_1, size * sizeof(float)));
  CUDACHECK(hipMalloc(&recvbuff_1, size * sizeof(float)));
  
  CUDACHECK(hipMemset(sendbuff_0, 0, size * sizeof(float)));
  CUDACHECK(hipMemset(recvbuff_0, 0, size * sizeof(float)));
  CUDACHECK(hipMemset(sendbuff_1, 0, size * sizeof(float)));
  CUDACHECK(hipMemset(recvbuff_1, 0, size * sizeof(float)));
 
  CUDACHECK(hipStreamCreate(&s_0));
  CUDACHECK(hipStreamCreate(&s_1));


  //initializing NCCL
  NCCLCHECK(ncclCommInitRank(&comm_0, nRanks, id_0, myRank));
  NCCLCHECK(ncclCommInitRank(&comm_1, nRanks, id_1, myRank));


  //communicating using NCCL
  NCCLCHECK(ncclAllReduce((const void*)sendbuff_0, (void*)recvbuff_0, size, ncclFloat, ncclSum, comm_0, s_0));
  // sleep(2);
  CUDACHECK(hipStreamSynchronize(s_0));

  printf("[MPI Rank %d] Success \n", myRank);

  NCCLCHECK(ncclAllReduce((const void*)sendbuff_1, (void*)recvbuff_1, size, ncclFloat, ncclSum, comm_1, s_1));

  // //completing NCCL operation by synchronizing on the CUDA stream
  // CUDACHECK(hipStreamSynchronize(s_0));
  CUDACHECK(hipStreamSynchronize(s_1));
  printf("[Rank MPI %d] Success \n", myRank);


  //free device buffers
  CUDACHECK(hipFree(sendbuff_0));
  CUDACHECK(hipFree(recvbuff_0));
  CUDACHECK(hipFree(sendbuff_1));
  CUDACHECK(hipFree(recvbuff_1));

  //finalizing NCCL
  ncclCommDestroy(comm_0);
  ncclCommDestroy(comm_1);


  //finalizing MPI
  MPICHECK(MPI_Finalize());


  // printf("[MPI Rank %d] Success \n", myRank);
  
  return 0;
}
